#include <hip/hip_runtime.h>
#include <hipblas.h>


#include <stdio.h>
#include <stdlib.h>

inline void checkCudaStatus(hipError_t status) {
    if (status != hipSuccess) {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

int main(int argc, char **argv)
{
    hipblasHandle_t cublasHandle;
    checkCublasStatus(hipblasCreate(&cublasHandle));

    printf("cublas initialized\n");

	return 0;
}