#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include <hipblaslt.h>

inline void checkCudaStatus(hipError_t status)
{
    if (status != hipSuccess)
    {
        printf("cuda API failed with status %d: %s\n", status, hipGetErrorString(status));
        exit(-1);
    }
}

inline void checkCublasStatus(hipblasStatus_t status)
{
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        printf("cuBLAS API failed with status %d\n", status);
        exit(-1);
    }
}

__global__ void gpuRandFunc(float* arr, uint32_t size, uint32_t seed1, uint32_t seed2)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        uint32_t Hash = idx;

        Hash ^= seed1;
        Hash *= 0xBAC57D37;
        Hash ^= seed2;
        Hash *= 0x24F66AC9;

        arr[idx] = int32_t(Hash) * 0.0000000004656612875245796f;
    }
}

__global__ void yess()
{

}

struct GpuRand {
    uint32_t seed1, seed2;

    GpuRand() {
        seed1 = 0xE621B963;
        seed2 = 0x6053653F;

        printf("Seed1: %u\n", seed1);
        printf("Seed2: %u\n\n", seed2);
    }

    void Rand(float* arr, uint32_t size) {
        seed1 ^= seed2;
        seed1 *= 0xBAC57D37;
        seed2 ^= seed1;
        seed2 *= 0x24F66AC9;

        gpuRandFunc <<<ceil(0.0009765625f * size), 1024>>> (arr, size, seed1, seed2);
    }
};

int main()
{
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;

    float alpha = 1.0;
    float beta = 0.0;

    size_t m = 1024;
    size_t n = 1024;
    size_t k = 1024;
    size_t N = 32;

    size_t lda = 1024;
    size_t ldb = 1024;
    size_t ldc = 1024;

    float* Adev, * Bdev, * Cdev, * biasDev;

    size_t workspaceSize = 1024 * 1024 * 4;
    void* workspace = NULL;

    int returnedResults = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult;
    
    hipblasLtHandle_t ltHandle;

    checkCublasStatus(hipblasLtCreate(&ltHandle));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Adev), m * k * N * sizeof(float)));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Bdev), n * k * N * sizeof(float)));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&Cdev), m * n * N * sizeof(float)));
    checkCudaStatus(hipMalloc(reinterpret_cast<void**>(&biasDev), m * N * sizeof(float)));
    checkCudaStatus(hipMalloc(&workspace, workspaceSize));

    GpuRand rand;
    rand.Rand(Adev, m * k * N);
    rand.Rand(Bdev, n * k * N);
    rand.Rand(biasDev, m * N);



    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa)));
    
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_32F, m, n, ldc));
    checkCublasStatus(hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_32F, m, n, ldc));

    checkCublasStatus(hipblasLtMatmulPreferenceCreate(&preference));
    checkCublasStatus(hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize)));

    checkCublasStatus(hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults));

    if (returnedResults == 0) checkCublasStatus(HIPBLAS_STATUS_NOT_SUPPORTED);

    checkCublasStatus(hipblasLtMatmul(ltHandle,
        operationDesc,
        &alpha,
        Adev,
        Adesc,
        Bdev,
        Bdesc,
        &beta,
        biasDev,
        Cdesc,
        Cdev,
        Ddesc,
        &heuristicResult.algo,
        workspace,
        workspaceSize,
        0));

    return 0;
}