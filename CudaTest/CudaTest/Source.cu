#include "Header.cuh"

//template <typename InType, typename OutType = InType, typename ComputeType = OutType>

int main() {
    hipblasLtHandle_t ltHandle;
    checkCublasStatus(hipblasLtCreate(&ltHandle));

    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL;

    hipblasOperation_t transa = HIPBLAS_OP_N, transb = HIPBLAS_OP_N;

    checkCublasStatus(hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
    checkCublasStatus(hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

    printf("JK");
    return 0;
}